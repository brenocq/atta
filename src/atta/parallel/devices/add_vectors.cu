#include "hip/hip_runtime.h"
#include "add_vectors.h"
#include <hip/hip_runtime_api.h>
#include <stdio.h>

__global__ void add_vectors_kernel(float* a, float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void add_vectors(float* a, float* b, float* c, int n) {
    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;

    hipMalloc((void**)&dev_a, n * sizeof(float));
    hipMalloc((void**)&dev_b, n * sizeof(float));
    hipMalloc((void**)&dev_c, n * sizeof(float));

    hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    add_vectors_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, n);

    hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
