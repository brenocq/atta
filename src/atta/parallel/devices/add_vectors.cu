#include "hip/hip_runtime.h"
#include "add_vectors.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add_vectors_kernel(float* a, float* b, float* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

void add_vectors(float* a, float* b, float* c, int n) {
    hipFuncAttributes attr;
    hipError_t err = hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(add_vectors_kernel));
    if (err != hipSuccess) {
        printf("hipFuncGetAttributes failed: %s\n", hipGetErrorString(err));
        return;
    }
    printf("Maximum threads per block: %d\n", attr.maxThreadsPerBlock);
    printf("Number of registers per thread: %d\n", attr.numRegs);

    float* dev_a = 0;
    float* dev_b = 0;
    float* dev_c = 0;

    hipMalloc((void**)&dev_a, n * sizeof(float));
    hipMalloc((void**)&dev_b, n * sizeof(float));
    hipMalloc((void**)&dev_c, n * sizeof(float));

    hipMemcpy(dev_a, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    add_vectors_kernel<<<blocksPerGrid, threadsPerBlock>>>(dev_a, dev_b, dev_c, n);

    hipMemcpy(c, dev_c, n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}
